#include "hip/hip_runtime.h"
/*
 * L3.cu
 *
 *  Created on: Mar 12, 2015
 *      Author: mvc
 */

#include "L3.h"

L3::L3() {
	// TODO Auto-generated constructor stub

}

L3::~L3() {
	// TODO Auto-generated destructor stub
}

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <errno.h>
#include <unistd.h>
#include <dirent.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

using namespace std;

// Define constant parameters
#define cfa_size      4
#define num_filter    5
#define patch_size    9
#define border_size   4
#define image_width   720
#define image_height  1280
#define voltage_max   0.9734
#define lum_list_size 20
#define num_out       3
#define low           0.95
#define high          1.15
#define sat_levels    19

#define num_frames    100

/* Cuda function - L3Render
 
 Compute mean for each channel
 
 Inputs:
 out_image       - pre-allocated space for output (xyz) image
 image           - original image
 cfa             - cfa pattern, should be 0-indexed
 lum_list        - luminance list
 sat_list        - saturation list
 flat_filters    - filters for flat regions
 texture_filters - filters for texture regions
 */
__global__ 
void L3Render(float* const out_image,
              float  const * const image,
              float  const * const lum_list,
              float  const * const sat_list,
              float  const * const flat_filters,
              float  const * const texture_filters,
              float  const * const flat_threshold_list)
{
    // Find pixel position
    const int row = blockIdx.x;
    const int col = threadIdx.x;
	const size_t out_pixel_index = num_out*(row + col * image_height);
    
    // Check pixel range
    if ((row < border_size) ||
        (row >= image_height - border_size) ||
        (col < border_size) ||
        (col >= image_width - border_size)){
			return;
	}

    // Compute patch type
    const unsigned short patch_type[] = {row % cfa_size, col % cfa_size};          // patch type
	const unsigned short cfa[] = {1, 0, 1, 3, 4, 1, 2, 1, 1, 3, 1, 0, 2, 1, 4, 1}; // cfa pattern

	// Store patch data in image to local memory
	float patch_data[patch_size*patch_size];

	
    
    // Compute mean for each channel
    float channel_mean[num_filter] = {0.0};
    unsigned short channel_count[num_filter] = {0};
    unsigned short cfa_index[patch_size * patch_size];
    
	unsigned short index = 0;
	unsigned short col_index = (col - border_size) % cfa_size;
	unsigned short row_index = (row - border_size) % cfa_size;
	size_t pixel_index = (row - border_size) + (col - border_size)*image_height;
	bool is_sat[num_filter] = {false};

    for (short jj = -border_size; jj <= border_size; jj++){
		unsigned short j_index = col_index * cfa_size;
		unsigned short i_index = row_index;
        for (short ii = -border_size; ii <= border_size; ii++){
            cfa_index[index] = i_index + j_index;
            channel_count[cfa[cfa_index[index]]] += 1;
			patch_data[index] = image[pixel_index];
			channel_mean[cfa[cfa_index[index]]] += patch_data[index];
			is_sat[cfa[cfa_index[index]]] |= (patch_data[index] > voltage_max);
			index++; pixel_index++; i_index = (i_index + 1) % cfa_size;
        }
		pixel_index += image_height-2*border_size-1;
		col_index = (col_index + 1) % cfa_size;
    }

    
    // Compute channel mean luminance
	// Channel mean luminance is defined as the mean of channel_mean
    float lum_mean = 0;
    for (int ii = 0; ii < num_filter; ii++) {
        channel_mean[ii] /= channel_count[ii];
        lum_mean += channel_mean[ii];
    }
    lum_mean /= num_filter;
    
    // Convert luminance to luminance index
    // Binary search might be faster for large nubmer of luminance levels
	// But this difference can be ignored when we have only around 20 luminance levels
    unsigned short lum_index = lum_list_size - 1;
    for (int ii = 0; ii < lum_list_size; ii++) {
        if (lum_mean < lum_list[ii]) {
            lum_index = ii;
            break;
        }
    }
    
    // Compute saturation type
    unsigned short sat_type = 0; // sat_type is the encoded saturation type
    unsigned short sat_index;    // sat_index is the number found with sat_list
    // const unsigned short sat_list_size = (1 << num_filter);
    for (int ii = num_filter - 1; ii >= 0; ii --)
        sat_type = (sat_type << 1) + is_sat[ii]; // (channel_mean[ii] > voltage_max);
    
	const float *cur_sat_list = sat_list + ((patch_type[1] * cfa_size + patch_type[0]) << num_filter);
    sat_index = cur_sat_list[sat_type];
    
    // Find nearest sat_type for missing ones
	const unsigned short sat_list_size = (1 << num_filter);
    if (sat_index == 0){
        float min_cost = 10000; // Init min cost to some arbitrarily large value
        for (int ii = 0; ii < sat_list_size; ii++) {
			if (cur_sat_list[ii] != 0){
				// compute cost
				float cur_cost = 0;
				unsigned short sat_diff = (ii ^ sat_type);
				for (int jj = 0; sat_diff > 0; jj++) {
					if ((sat_diff & 1) > 0)
						cur_cost += fabsf(channel_mean[jj] - voltage_max);
					sat_diff = (sat_diff >> 1);
				}
				if (cur_cost < min_cost) {
					min_cost = cur_cost;
					sat_index = cur_sat_list[ii];
				}
			}
		}
    }
    sat_index--; // make sat_index 0-indexed
    
    // Compute image contrast
    // Assume image_contrast array has been allocated as zeros
    float image_contrast = 0;
	for (index = 0; index < patch_size * patch_size; index++)
		image_contrast += fabsf(patch_data[index] - channel_mean[cfa[cfa_index[index]]]);
	image_contrast /= (patch_size * patch_size);
    
    // Determine flat or texture
    const int threshold_index  = ((sat_index * lum_list_size + lum_index) * cfa_size + patch_type[1]) * cfa_size + patch_type[0];
    const float flat_threshold = flat_threshold_list[threshold_index];

	// Apply filter to patch
	const float *filter;
	float out_data[num_out] = {0};
	unsigned short filter_index;
	unsigned int filter_offset = threshold_index * num_out * patch_size * patch_size;
    if (image_contrast < flat_threshold * low) { // flat region
        filter = flat_filters + filter_offset;	
		for (index = 0, filter_index = 0; index < patch_size * patch_size; index++, filter_index += 3){
			out_data[0] += patch_data[index] * filter[filter_index];
			out_data[1] += patch_data[index] * filter[filter_index+1];
            out_data[2] += patch_data[index] * filter[filter_index+2];
		}
    }
    else if (image_contrast > flat_threshold * high) { // texture region
        filter = texture_filters + filter_offset;
		for (index = 0, filter_index = 0; index < patch_size * patch_size; index++, filter_index += 3){
			out_data[0] += patch_data[index] * filter[filter_index];
			out_data[1] += patch_data[index] * filter[filter_index+1];
            out_data[2] += patch_data[index] * filter[filter_index+2];
		}
    }
    else { // transition region
        const float weights = (image_contrast / flat_threshold - low) / (high - low);
        filter = flat_filters + filter_offset;
        const float* filter_texture = texture_filters + filter_offset;
		for (index = 0, filter_index = 0; index < patch_size * patch_size; index ++, filter_index += 3){
			out_data[0] += patch_data[index] * (filter[filter_index]   * weights + filter_texture[filter_index] * (1 - weights));
            out_data[1] += patch_data[index] * (filter[filter_index+1] * weights + filter_texture[filter_index+1] * (1 - weights));
			out_data[2] += patch_data[index] * (filter[filter_index+2] * weights + filter_texture[filter_index+2] * (1 - weights));
		}
    }
	out_image[out_pixel_index] = out_data[0];
	out_image[out_pixel_index + 1] = out_data[1];
	out_image[out_pixel_index + 2] = out_data[2];
}

// Main routine
int L3::L3_main(void)
{
	LOGD("hello");
    // Init parameters
    float * out_image, * out_image_d; // pointer to rendered image
    float * image, * image_d; // pointer to input raw image data
    float  * lum_list, * lum_list_d; // pointer to luminance list
    float  * sat_list, * sat_list_d; // pointer to saturation list
    float  * flat_filters, * flat_filters_d; // pointer to filters in flat region
    float  * texture_filters, * texture_filters_d; // pointer to filters in texture region
    float  * flat_threshold_list, * flat_threshold_list_d; // the list of thresholds of determining a patch is flat or not
    
    const unsigned short sat_list_size = (1 << num_filter)*cfa_size*cfa_size;
	const unsigned int flat_filters_size = num_out * patch_size * patch_size * lum_list_size * cfa_size * cfa_size * sat_levels;
    const unsigned int texture_filters_size = flat_filters_size;
	const unsigned int flat_threshold_list_size = lum_list_size*cfa_size*cfa_size*sat_levels;
    
	LOGD("hello 1");
    // Allocate spaces in main memory
    image = (float*)malloc(sizeof(float)*image_width*image_height);
    out_image = (float*) malloc(sizeof(float)*image_height*image_width*num_out);
    lum_list = (float*) malloc(sizeof(float)*lum_list_size);
    sat_list = (float*) malloc(sizeof(float)*sat_list_size);
    flat_filters = (float*) malloc(sizeof(float) * flat_filters_size);
    texture_filters = (float*)malloc(sizeof(float)* texture_filters_size);
	flat_threshold_list = (float*)malloc(sizeof(float)*flat_threshold_list_size);
    
	LOGD("hello 2");
    // Load data from files
    FILE* pf;
   
    pf = fopen("/sdcard/cudadata/lum_list.dat", "rb"); // luminance list
   
    	
	LOGD("hello 3");
    fread(lum_list, sizeof(float), lum_list_size, pf);
    
	LOGD("hello 4");
	fclose(pf);
    
    pf = fopen("/sdcard/cudadata/sat_list.dat", "rb"); // saturation list
    fread(sat_list, sizeof(float), sat_list_size, pf);
    fclose(pf);
    
    pf = fopen("/sdcard/cudadata/flat_filters.dat", "rb"); //flat filters
    fread(flat_filters, sizeof(float), flat_filters_size, pf);
    fclose(pf);
    
    pf = fopen("/sdcard/cudadata/texture_filters.dat", "rb"); // texture filters
    fread(texture_filters, sizeof(float), texture_filters_size, pf);
    fclose(pf);
    
    pf = fopen("/sdcard/cudadata/flat_threshold_list.dat", "rb"); // flat threshold list
	fread(flat_threshold_list, sizeof(float), flat_threshold_list_size, pf);
    fclose(pf);
    
    // Allocate spaces in GPU
    hipMalloc((void **) & out_image_d, sizeof(float)*image_width*image_height*num_out);
	hipMalloc((void **) & image_d, sizeof(float)*image_height * image_width);
    hipMalloc((void **) & lum_list_d, sizeof(float)*lum_list_size);
    hipMalloc((void **) & sat_list_d, sizeof(float)*sat_list_size);
    hipMalloc((void **) & flat_filters_d, sizeof(float)*flat_filters_size);
    hipMalloc((void **) & texture_filters_d, sizeof(float)*texture_filters_size);
	hipMalloc((void **) & flat_threshold_list_d, sizeof(float)*flat_threshold_list_size);
    
    // Copy data to GPU
    hipMemcpy(lum_list_d, lum_list, sizeof(float)*lum_list_size, hipMemcpyHostToDevice);
    hipMemcpy(sat_list_d, sat_list, sizeof(float)*sat_list_size, hipMemcpyHostToDevice);
    hipMemcpy(flat_filters_d, flat_filters, sizeof(float)*flat_filters_size, hipMemcpyHostToDevice);
    hipMemcpy(texture_filters_d, texture_filters, sizeof(float)*texture_filters_size, hipMemcpyHostToDevice);
	hipMemcpy(flat_threshold_list_d, flat_threshold_list, sizeof(float)*flat_threshold_list_size, hipMemcpyHostToDevice);

	char *fName = new char[100];
	struct timeval tm1, tm2;
	
	for (int fIndex = 0; fIndex < num_frames; fIndex ++ ){
		//Runtime including IO
		//gettimeofday(&tm1, NULL);
		//LOGD("frame # %d", fIndex);
		// show debug info
		printf("Processing frame %d...\n", fIndex);

		// Load image
		// sprintf(fName, "./video/output_%07d.dat", fIndex);
		sprintf(fName, "/sdcard/cudadata/raw_image.dat", fIndex);
		pf = fopen(fName, "rb"); // image raw data
		fread(image, sizeof(float), image_width * image_height, pf);
		fclose(pf);
		
		hipMemcpy(image_d, image, sizeof(float)*image_height*image_width, hipMemcpyHostToDevice);
		hipMemset(out_image_d, 0, image_width*image_height*num_out*sizeof(float));
    
    	gettimeofday(&tm1, NULL);
		
		// Do computation in GPU
		L3Render<<<image_height, image_width>>>(out_image_d, image_d, lum_list_d, sat_list_d, flat_filters_d, texture_filters_d, flat_threshold_list_d);
    	
		// Copy back to main memory
		hipMemcpy(out_image, out_image_d, sizeof(float)*image_width*image_height*num_out, hipMemcpyDeviceToHost);
    	gettimeofday(&tm2, NULL);
		unsigned long long t = 1000000 * (tm2.tv_sec - tm1.tv_sec) + (tm2.tv_usec - tm1.tv_usec);
		LOGD("time lapse: %llu", t);
		
    	
		// Write rendered image to file
		// sprintf(fName, "./video_out/frame%07d.dat", fIndex);
		sprintf(fName, "/sdcard/cudadata/raw_image.dat", fIndex);
		pf = fopen(fName, "wb");
		fwrite(out_image, sizeof(float), image_height * image_width * num_out, pf);
		fclose(pf);
		
		//runtime including IO
		/*gettimeofday(&tm2, NULL);
		unsigned long long t = 1000 * (tm2.tv_sec - tm1.tv_sec) + (tm2.tv_usec - tm1.tv_usec) / 1000;
		LOGD("time lapse: %llu", t);*/
	}
	LOGD("done");
	
    // Cleanup and return
    free(out_image); hipFree(out_image_d);
    free(image); hipFree(image_d);
    free(lum_list); hipFree(lum_list_d);
    free(sat_list); hipFree(sat_list_d);
    free(flat_filters); hipFree(flat_filters_d);
    free(texture_filters); hipFree(texture_filters_d);
    free(flat_threshold_list); hipFree(flat_threshold_list_d);
    
    LOGD("done 4");
	//hipDeviceReset();
	LOGD("blah 4");
	
    return 0;
}

